#include "cuda_utils.h"

#include <stdio.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

namespace vcsmc {

bool InitializeCuda() {
  // Initialize CUDA, use first device.
  int cuda_device_count = 0;
  hipError_t cuda_error = hipGetDeviceCount(&cuda_device_count);
  if (cuda_error != hipSuccess) {
    fprintf(stderr, "CUDA error on device enumeration.\n");
    fprintf(stderr,"%s: %s\n", hipGetErrorName(cuda_error),
                               hipGetErrorString(cuda_error));
    return false;
  } else if (!cuda_device_count) {
    fprintf(stderr, "unable to find CUDA device.\n");
    return false;
  }
  // Ensure synchronization behavior consistent with our needs.
  hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
  hipSetDevice(0);
  hipDeviceProp_t device_props;
  hipGetDeviceProperties(&device_props, 0);
  printf("CUDA Device 0: \"%s\" with compute capability %d.%d.\n",
      device_props.name, device_props.major, device_props.minor);
  return true;
}

}  // namespace vcsmc
